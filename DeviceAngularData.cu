#include "hip/hip_runtime.h"
#include "AngularData.h"

#include "common.cuh"

#include <stdio.h>

#ifndef _
#define _(x) do { \
	if ((x) != hipSuccess) { \
	fprintf(stderr, "File %s line %d, %s failed with error `%s'\n", __FILE__, __LINE__, #x, hipGetErrorString(hipGetLastError())); \
	fflush(stderr); } \
} while (0)
#endif

__global__ void copy_omega(REAL *dst, REAL *src, idx slm) {
	idx aslm = blockDim.x;
	#pragma unroll
	for (int s = 0; s < 3; s++) {
		if ((threadIdx.x < slm) && (blockIdx.x < slm))
			dst[s*aslm*aslm + aslm * blockIdx.x + threadIdx.x] = src[s*slm*slm + slm * blockIdx.x + threadIdx.x];
		else
			dst[s*aslm*aslm + aslm * blockIdx.x + threadIdx.x] = 0;
	}
}

__global__ void copy_omega_pos(idx *dst, idx *src, idx slm) {
	idx aslm = blockDim.x;
	#pragma unroll
	for (int s = 0; s < 3; s++) {
		if ((threadIdx.x < slm) && (blockIdx.x < slm))
			dst[s*aslm*aslm + aslm * blockIdx.x + threadIdx.x] = src[s*slm*slm + slm * blockIdx.x + threadIdx.x];
		else
			dst[s*aslm*aslm + aslm * blockIdx.x + threadIdx.x] = 0;
	}
}

__global__ void copy_On(REAL *dst, REAL *src, idx slm) {
	idx aslm = blockDim.x;
	if ((threadIdx.x < slm) && (blockIdx.x < slm))
		dst[aslm * blockIdx.x + threadIdx.x] = src[slm * blockIdx.x + threadIdx.x];
	else
		dst[aslm * blockIdx.x + threadIdx.x] = 0;
}

DeviceAngularData::DeviceAngularData(const AngularData &host) {
	slm = host.slm;
	aslm = align_power(host.slm, COALESCED_NUM(REAL));
	_(hipMalloc((void **)&omega, 3*aslm*aslm*sizeof(REAL)));
	_(hipMalloc((void **)&omega_pos, 3*aslm*aslm*sizeof(idx)));
	_(hipMalloc((void **)&Ox, aslm*aslm*sizeof(REAL)));
	_(hipMalloc((void **)&Oy, aslm*aslm*sizeof(REAL)));
	_(hipMalloc((void **)&Oz, aslm*aslm*sizeof(REAL)));

	printf("DeviceAngularData:\n");
	printf("\tomega     = %p\n", omega);
	printf("\tomega_pos = %p\n", omega_pos);
	printf("\tOx        = %p\n", Ox);
	printf("\tOy        = %p\n", Oy);
	printf("\tOz        = %p\n", Oz);

	void *tmp;
	dim3 block;
	_(hipMalloc((void **)&tmp, 3*aslm*aslm*sizeof(REAL)));

	_(hipMemcpy(tmp, host.omega, 3*slm*slm*sizeof(REAL), hipMemcpyHostToDevice));
	copy_omega<<<aslm,aslm>>>(omega, (REAL *)tmp, slm);

	_(hipMemcpy(tmp, host.omega_pos, 3*slm*slm*sizeof(idx), hipMemcpyHostToDevice));
	copy_omega_pos<<<aslm,aslm>>>(omega_pos, (idx *)tmp, slm);

	_(hipMemcpy(tmp, host.Ox, slm*slm*sizeof(REAL), hipMemcpyHostToDevice));
	copy_On<<<aslm,aslm>>>(Ox, (REAL *)tmp, slm);

	_(hipMemcpy(tmp, host.Oy, slm*slm*sizeof(REAL), hipMemcpyHostToDevice));
	copy_On<<<aslm,aslm>>>(Oy, (REAL *)tmp, slm);

	_(hipMemcpy(tmp, host.Oz, slm*slm*sizeof(REAL), hipMemcpyHostToDevice));
	copy_On<<<aslm,aslm>>>(Oz, (REAL *)tmp, slm);

	_(hipFree(tmp));
}

DeviceAngularData::~DeviceAngularData() {
	_(hipFree(omega));
	_(hipFree(omega_pos));
	_(hipFree(Ox));
	_(hipFree(Oy));
	_(hipFree(Oz));
}
