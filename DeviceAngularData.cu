#include "hip/hip_runtime.h"
#include "AngularData.h"

#include "common.cuh"

#include <stdio.h>

__global__ void copy_omega(REAL *dst, REAL *src, idx slm) {
	idx aslm = blockDim.x;
	#pragma unroll
	for (int s = 0; s < 3; s++) {
		if ((threadIdx.x < slm) && (blockIdx.x < slm))
			dst[s*aslm*aslm + aslm * blockIdx.x + threadIdx.x] = src[s*slm*slm + slm * blockIdx.x + threadIdx.x];
		else
			dst[s*aslm*aslm + aslm * blockIdx.x + threadIdx.x] = 0;
	}
}

__global__ void copy_omega_pos(idx *dst, idx *src, idx slm) {
	idx aslm = blockDim.x;
	#pragma unroll
	for (int s = 0; s < 3; s++) {
		if ((threadIdx.x < slm) && (blockIdx.x < slm))
			dst[s*aslm*aslm + aslm * blockIdx.x + threadIdx.x] = src[s*slm*slm + slm * blockIdx.x + threadIdx.x];
		else
			dst[s*aslm*aslm + aslm * blockIdx.x + threadIdx.x] = 0;
	}
}

__global__ void copy_On(REAL *dst, REAL *src, idx slm) {
	idx aslm = blockDim.x;
	if ((threadIdx.x < slm) && (blockIdx.x < slm))
		dst[aslm * blockIdx.x + threadIdx.x] = src[slm * blockIdx.x + threadIdx.x];
	else
		dst[aslm * blockIdx.x + threadIdx.x] = 0;
}

DeviceAngularData::DeviceAngularData(const AngularData &host) {
	slm = host.slm;
	aslm = align_power(host.slm, COALESCED_NUM(REAL));
	hipMalloc((void **)&omega, 3*aslm*aslm*sizeof(REAL));
	hipMalloc((void **)&omega_pos, 3*aslm*aslm*sizeof(idx));
	hipMalloc((void **)&Ox, aslm*aslm*sizeof(REAL));
	hipMalloc((void **)&Oy, aslm*aslm*sizeof(REAL));
	hipMalloc((void **)&Oz, aslm*aslm*sizeof(REAL));

	printf("DeviceAngularData:\n");
	printf("\tomega     = %p\n", omega);
	printf("\tomega_pos = %p\n", omega_pos);
	printf("\tOx        = %p\n", Ox);
	printf("\tOy        = %p\n", Oy);
	printf("\tOz        = %p\n", Oz);

	void *tmp;
	dim3 block;
	hipMalloc((void **)&tmp, 3*aslm*aslm*sizeof(REAL));

	hipMemcpy(tmp, host.omega, 3*slm*slm*sizeof(REAL), hipMemcpyHostToDevice);
	copy_omega<<<aslm,aslm>>>(omega, (REAL *)tmp, slm);

	hipMemcpy(tmp, host.omega_pos, 3*slm*slm*sizeof(idx), hipMemcpyHostToDevice);
	copy_omega_pos<<<aslm,aslm>>>(omega_pos, (idx *)tmp, slm);

	hipMemcpy(tmp, host.Ox, slm*slm*sizeof(REAL), hipMemcpyHostToDevice);
	copy_On<<<aslm,aslm>>>(Ox, (REAL *)tmp, slm);

	hipMemcpy(tmp, host.Oy, slm*slm*sizeof(REAL), hipMemcpyHostToDevice);
	copy_On<<<aslm,aslm>>>(Oy, (REAL *)tmp, slm);

	hipMemcpy(tmp, host.Oz, slm*slm*sizeof(REAL), hipMemcpyHostToDevice);
	copy_On<<<aslm,aslm>>>(Oz, (REAL *)tmp, slm);

	hipFree(tmp);
}

DeviceAngularData::~DeviceAngularData() {
	hipFree(omega);
	hipFree(omega_pos);
	hipFree(Ox);
	hipFree(Oy);
	hipFree(Oz);
}
