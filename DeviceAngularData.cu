#include "AngularData.h"

#include "common.cuh"

#include <stdio.h>

#ifndef _
#define _(x) do { \
	if ((x) != hipSuccess) { \
	fprintf(stderr, "File %s line %d, %s failed with error `%s'\n", __FILE__, __LINE__, #x, hipGetErrorString(hipGetLastError())); \
	fflush(stderr); } \
} while (0)
#endif

DeviceAngularData::DeviceAngularData(const AngularData &host) {
	slm = host.slm;
	aslm = align_power(host.slm, COALESCED_NUM(REAL));
	_(hipMalloc((void **)&omega, 3*aslm*aslm*sizeof(REAL)));
	_(hipMalloc((void **)&omega_pos, 3*aslm*aslm*sizeof(idx)));
	_(hipMalloc((void **)&Ox, aslm*aslm*sizeof(REAL)));
	_(hipMalloc((void **)&Oy, aslm*aslm*sizeof(REAL)));
	_(hipMalloc((void **)&Oz, aslm*aslm*sizeof(REAL)));

	printf("DeviceAngularData:\n");
	printf("\tomega     = %p\n", omega);
	printf("\tomega_pos = %p\n", omega_pos);
	printf("\tOx        = %p\n", Ox);
	printf("\tOy        = %p\n", Oy);
	printf("\tOz        = %p\n", Oz);

	dim3 block;
	REAL *omega_aligned = new REAL[3*aslm*aslm];
	idx *omega_pos_aligned = new idx[3*aslm*aslm];
	idx *Ox_aligned = new idx[aslm*aslm];
	idx *Oy_aligned = new idx[aslm*aslm];
	idx *Oz_aligned = new idx[aslm*aslm];

	for (idx i=0; i < aslm; i++)
		for (idx j=0; j < aslm; j++) 
			if (i < slm && j < slm) {
				omega_aligned[0*aslm*aslm + i*aslm + j] = host.omega[0*slm*slm + i*slm + j];
				omega_aligned[1*aslm*aslm + i*aslm + j] = host.omega[1*slm*slm + i*slm + j];
				omega_aligned[2*aslm*aslm + i*aslm + j] = host.omega[2*slm*slm + i*slm + j];
				omega_pos_aligned[0*aslm*aslm + i*aslm + j] = host.omega_pos[0*slm*slm + i*slm + j];
				omega_pos_aligned[1*aslm*aslm + i*aslm + j] = host.omega_pos[1*slm*slm + i*slm + j];
				omega_pos_aligned[2*aslm*aslm + i*aslm + j] = host.omega_pos[2*slm*slm + i*slm + j];
				Ox_aligned[i*aslm + j] = host.Ox[i*slm + j];
				Oy_aligned[i*aslm + j] = host.Oy[i*slm + j];
				Oz_aligned[i*aslm + j] = host.Oz[i*slm + j];
			} else {
				omega_aligned[0*aslm*aslm + i*aslm + j] = 0;
				omega_aligned[1*aslm*aslm + i*aslm + j] = 0;
				omega_aligned[2*aslm*aslm + i*aslm + j] = 0;
				omega_pos_aligned[0*aslm*aslm + i*aslm + j] = 0;
				omega_pos_aligned[1*aslm*aslm + i*aslm + j] = 0;
				omega_pos_aligned[2*aslm*aslm + i*aslm + j] = 0;
				Ox_aligned[i*aslm + j] = 0;
				Oy_aligned[i*aslm + j] = 0;
				Oz_aligned[i*aslm + j] = 0;
			}
		
	_(hipMemcpy(omega, omega_aligned, 3*aslm*aslm*sizeof(REAL), hipMemcpyHostToDevice));
	_(hipMemcpy(omega_pos, omega_pos_aligned, 3*aslm*aslm*sizeof(idx), hipMemcpyHostToDevice));
	_(hipMemcpy(Ox, Ox_aligned, aslm*aslm*sizeof(REAL), hipMemcpyHostToDevice));
	_(hipMemcpy(Oy, Oy_aligned, aslm*aslm*sizeof(REAL), hipMemcpyHostToDevice));
	_(hipMemcpy(Oz, Oz_aligned, aslm*aslm*sizeof(REAL), hipMemcpyHostToDevice));
}

DeviceAngularData::~DeviceAngularData() {
	_(hipFree(omega));
	_(hipFree(omega_pos));
	_(hipFree(Ox));
	_(hipFree(Oy));
	_(hipFree(Oz));
}
