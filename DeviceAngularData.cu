#include "hip/hip_runtime.h"
#include "AngularData.h"

#include "common.cuh"

__global__ copy_omega(REAL *dst, REAL *src, idx slm) {
	idx aslm = blockDim.x;
	#pragma unroll
	for (int s = 0; s < 3; s++) {
		if ((threadIdx.x < slm) && (blockIdx.x < slm))
			dst[s*aslm*aslm + aslm * blockIdx.x + threadIdx.x] = src[s*slm*slm + slm * blockIdx.x + threadIdx.x];
		else
			dst[s*aslm*aslm + aslm * blockIdx.x + threadIdx.x] = 0;
	}
}

DeviceAngularData::DeviceAngularData(const AngularData &host) {
	slm = align_power(host.slm, COALESCED_NUM(REAL));
	hipMalloc(omega, 3*slm*slm*sizeof(REAL));
	hipMalloc(omega_pos, 3*slm*slm*sizeof(idx));
	hipMalloc(Ox, slm*slm*sizeof(REAL));
	hipMalloc(Oy, slm*slm*sizeof(REAL));
	hipMalloc(Oz, slm*slm*sizeof(REAL));

	void *tmp;
	dim3 block;
	hipMalloc(tmp, 3*slm*slm*sizeof(REAL));

	hipMemcpy(tmp, host.omega, hipMemcpyHostToDevice);
	copy_omega<<<slm,slm>>>(omega, (REAL *)tmp, host.slm);

	hipMemcpy(tmp, host.omega_pos, hipMemcpyHostToDevice);
	copy_omega_pos<<<slm,slm>>>(omega, (idx *)tmp, host.slm);

	hipMemcpy(tmp, host.Ox, hipMemcpyHostToDevice);
	copy_On<<<slm,slm>>>(Ox, (REAL *)tmp, host.slm);

	hipMemcpy(tmp, host.Oy, hipMemcpyHostToDevice);
	copy_On<<<slm,slm>>>(Oy, (REAL *)tmp, host.slm);

	hipMemcpy(tmp, host.Oz, hipMemcpyHostToDevice);
	copy_On<<<slm,slm>>>(Oz, (REAL *)tmp, host.slm);

	hipFree(tmp);
}

DeviceAngularData::~DeviceAngularData() {
	hipFree(omega);
	hipFree(omega_pos);
	hipFree(Ox);
	hipFree(Oy);
	hipFree(Oz);
}
