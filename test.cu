#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void init(float *a) {
	int idx = threadIdx.x + (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x;
	a[idx] = idx;
}

__global__ void kernel(float *a) {
	int idx = threadIdx.x + (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x;
	a[idx] = sin(a[idx]);
}

int main() {
	const int N = 64*1024*1024;
	float *a;
	hipMalloc(&a, N*sizeof(float));
	printf("Error: %s\n", hipGetErrorString(hipGetLastError()));
	hipEvent_t start, end;
	hipEventCreate(&start);
	hipEventCreate(&end);
	printf("Error: %s\n", hipGetErrorString(hipGetLastError()));
	float time = 0.f;
	for (int bs = 16; bs <= 512; bs += 16) {
		dim3 grid(N/bs/1024, 1024);
		hipEventRecord(start, 0);
		init<<< grid, bs >>>(a);
		hipEventRecord(end, 0);
		hipEventSynchronize(end);
		printf("Error: %s\n", hipGetErrorString(hipGetLastError()));
		hipEventElapsedTime(&time, start, end);
		printf("[%10d] init : % 10.2fms\n", bs, time);
		hipEventRecord(start, 0);
		kernel<<< grid, bs >>>(a);
		hipEventRecord(end, 0);
		hipEventSynchronize(end);
		hipEventElapsedTime(&time, start, end);
		printf("[%10d] kern : % 10.2fms\n", bs, time);
	}
}
