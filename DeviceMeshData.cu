#include "hip/hip_runtime.h"
#include "MeshData.h"
#include <stdio.h>

#ifndef _
#define _(x) do { \
	if ((x) != hipSuccess) { \
	fprintf(stderr, "File %s line %d, %s failed with error `%s'\n", __FILE__, __LINE__, #x, hipGetErrorString(hipGetLastError())); \
	fflush(stderr); } \
} while (0)
#endif

DeviceMeshData::DeviceMeshData(const MeshData &host) {
	int nps = (int)(sqrt(host.nP)+0.5000001);
	nPlow = align_power(nps, 16);
	nPhigh = align_power(nps, 16);
	nP = host.nP;

	_(hipMalloc((void **)&tetstart, (nP+1)*sizeof(idx)));
	_(hipMalloc((void **)&tetidx, host.tetstart[host.nP]*sizeof(idx)));
	_(hipMalloc((void **)&tetpos, host.tetstart[host.nP]*sizeof(idx)));
	_(hipMalloc((void **)&mesh, host.nT*sizeof(tetrahedron)));

	_(hipMalloc((void **)&facestart, (nP+1)*sizeof(idx)));
	_(hipMalloc((void **)&faceidx, host.facestart[host.nP]*sizeof(idx)));
	_(hipMalloc((void **)&facepos, host.facestart[host.nP]*sizeof(idx)));
	_(hipMalloc((void **)&bnd, host.nF*sizeof(face)));

	printf("DeviceMeshData:\n");
	printf("\tmesh		= %p\n",mesh);
	printf("\ttetstart	= %p\n",tetstart);
	printf("\ttetidx	= %p\n",tetidx);
	printf("\ttetpos	= %p\n",tetpos);
	printf("\tbnd		= %p\n",bnd);
	printf("\tfacestart = %p\n",facestart);
	printf("\tfaceidx 	= %p\n",faceidx);
	printf("\tfacepos	= %p\n",facepos);

	_(hipMemcpy(tetstart, host.tetstart, (host.nP+1)*sizeof(idx), hipMemcpyHostToDevice));

	_(hipMemcpy(facestart, host.facestart, (host.nP+1)*sizeof(idx), hipMemcpyHostToDevice));

	_(hipMemcpy(tetidx, host.tetidx, host.tetstart[host.nP]*sizeof(idx), hipMemcpyHostToDevice));
	_(hipMemcpy(tetpos, host.tetpos, host.tetstart[host.nP]*sizeof(idx), hipMemcpyHostToDevice));
	_(hipMemcpy(mesh, host.mesh, host.nT*sizeof(tetrahedron), hipMemcpyHostToDevice));

	_(hipMemcpy(faceidx, host.faceidx, host.facestart[host.nP]*sizeof(idx), hipMemcpyHostToDevice));
	_(hipMemcpy(facepos, host.facepos, host.facestart[host.nP]*sizeof(idx), hipMemcpyHostToDevice));
	_(hipMemcpy(bnd, host.bnd, host.nF*sizeof(face), hipMemcpyHostToDevice));
}

DeviceMeshData::~DeviceMeshData() {
	printf("deleting\n");
	_(hipFree(tetstart));
	_(hipFree(tetidx));
	_(hipFree(tetpos));
	_(hipFree(mesh));

	_(hipFree(facestart));
	_(hipFree(faceidx));
	_(hipFree(facepos));
	_(hipFree(bnd));
}
