#include "hip/hip_runtime.h"
#include <MeshData.h>

__global__ void copy_start(idx *dst, idx *src, idx nP) {
	idx i = blockIdx.x + blockIdx.y * gridDim.x;
	
	dst[i] = src[i>nP ? nP : i];
}

DeviceMeshData::DeviceMeshData(const MeshData host) {
	int nps = (int)(sqrt(nP)+0.5000001);
	nPlow = align_power(nps, 16);
	nPhigh = align_power(nps, 16);
	int nP = nPlow * nPhigh;

	idx *tmp;
	hipMalloc((void **)&tmp, (host.nP+1)*sizeof(idx));

	hipMalloc((void **)&tetstart, (nP+1)*sizeof(idx));
	hipMalloc((void **)&tetidx, host.tetstart[host.nP]*sizeof(idx));
	hipMalloc((void **)&tetpos, host.tetstart[host.nP]*sizeof(idx));
	hipMalloc((void **)&mesh, host.nT*sizeof(tetrahedron));

	hipMalloc((void **)&facestart, (nP+1)*sizeof(idx));
	hipMalloc((void **)&faceidx, host.facestart[host.nP]*sizeof(idx));
	hipMalloc((void **)&facepos, host.facestart[host.nP]*sizeof(idx));
	hipMalloc((void **)&bnd, host.nF*sizeof(face));

	dim3 grid(nPlow, nPhigh);
	
	hipMemcpy(tmp, host.tetstart, (host.nP+1)*sizeof(idx), hipMemcpyHostToDevice);
	copy_start<<<grid,block>>>(tetstart, tmp, host.nP);

	hipMemcpy(tmp, host.facestart, (host.nP+1)*sizeof(idx), hipMemcpyHostToDevice);
	copy_start<<<grid,block>>>(facestart, tmp, host.nP);

	hipMemcpy(tetidx, host.tetidx, host.tetstart[host.nP]*sizeof(idx));
	hipMemcpy(tetpos, host.tetpos, host.tetstart[host.nP]*sizeof(idx));
	hipMemcpy(mesh, host.mesh, host.nT*sizeof(tetrahedron));

	hipMemcpy(faceidx, host.faceidx, host.facestart[host.nP]*sizeof(idx));
	hipMemcpy(facepos, host.facepos, host.facestart[host.nP]*sizeof(idx));
	hipMemcpy(bnd, host.bnd, host.nF*sizeof(face));

	hipFree(tmp);
}

DeviceMeshData::~DeviceMeshData() {
	hipFree(tetstart);
	hipFree(tetidx);
	hipFree(tetpos);
	hipFree(mesh);

	hipFree(facestart);
	hipFree(faceidx);
	hipFree(facepos);
	hipFree(bnd);
}
