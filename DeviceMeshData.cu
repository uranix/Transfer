#include "hip/hip_runtime.h"
#include <MeshData.h>

DeviceMeshData::DeviceMeshData(const MeshData &host) {
	int nps = (int)(sqrt(host.nP)+0.5000001);
	nPlow = align_power(nps, 16);
	nPhigh = align_power(nps, 16);
	nP = host.nP;

	hipMalloc((void **)&tetstart, (nP+1)*sizeof(idx));
	hipMalloc((void **)&tetidx, host.tetstart[host.nP]*sizeof(idx));
	hipMalloc((void **)&tetpos, host.tetstart[host.nP]*sizeof(idx));
	hipMalloc((void **)&mesh, host.nT*sizeof(tetrahedron));

	hipMalloc((void **)&facestart, (nP+1)*sizeof(idx));
	hipMalloc((void **)&faceidx, host.facestart[host.nP]*sizeof(idx));
	hipMalloc((void **)&facepos, host.facestart[host.nP]*sizeof(idx));
	hipMalloc((void **)&bnd, host.nF*sizeof(face));

	dim3 grid(nPlow, nPhigh);
	dim3 block(1);
	
	hipMemcpy(tetstart, host.tetstart, (host.nP+1)*sizeof(idx), hipMemcpyHostToDevice);

	hipMemcpy(facestart, host.facestart, (host.nP+1)*sizeof(idx), hipMemcpyHostToDevice);

	hipMemcpy(tetidx, host.tetidx, host.tetstart[host.nP]*sizeof(idx), hipMemcpyHostToDevice);
	hipMemcpy(tetpos, host.tetpos, host.tetstart[host.nP]*sizeof(idx), hipMemcpyHostToDevice);
	hipMemcpy(mesh, host.mesh, host.nT*sizeof(tetrahedron), hipMemcpyHostToDevice);

	hipMemcpy(faceidx, host.faceidx, host.facestart[host.nP]*sizeof(idx), hipMemcpyHostToDevice);
	hipMemcpy(facepos, host.facepos, host.facestart[host.nP]*sizeof(idx), hipMemcpyHostToDevice);
	hipMemcpy(bnd, host.bnd, host.nF*sizeof(face), hipMemcpyHostToDevice);
}

DeviceMeshData::~DeviceMeshData() {
	hipFree(tetstart);
	hipFree(tetidx);
	hipFree(tetpos);
	hipFree(mesh);

	hipFree(facestart);
	hipFree(faceidx);
	hipFree(facepos);
	hipFree(bnd);
}
