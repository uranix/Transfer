#include "kernels.h"
#include "kernels.cu"

void *deviceAlloc(size_t size) {
	void *ret;
	hipMalloc(&ret, size);
	return ret;
}

void deviceFree(void *mem) {
	hipFree(mem);
}

void computeRhs(const DeviceMeshDataRaw meshdata, const DeviceAngularDataRaw angdata, REAL *f, REAL *r) {
	dim3 grid(meshdata.nPlow, meshdata.nPhigh);
	dim3 block(angdata.aslm);
	volumePart<<<grid, block>>>(meshdata, angdata, f, r);
	surfacePart<<<grid, block>>>(meshdata, angdata, f, r);
}

