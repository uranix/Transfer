#include "hip/hip_runtime.h"
#include "kernels.h"
#include <stdio.h>
#include <math.h>

#ifndef _
#define _(x) do { \
	if ((x) != hipSuccess) { \
	fprintf(stderr, "File %s line %d, %s failed with error `%s'\n", __FILE__, __LINE__, #x, hipGetErrorString(hipGetLastError())); \
	fflush(stderr); } \
} while (0)
#endif

#include "kernels.cu"

void CudaContext::setDevice(int dev) {
	_(hipSetDevice(dev));
}

void *CudaContext::deviceAlloc(size_t size) {
	void *ret;
	_(hipMalloc(&ret, size));
	return ret;
}

void CudaContext::deviceFree(void *mem) {
	_(hipFree(mem));
}

void CudaContext::copyToDev(void *dst, void *src, size_t sz) {
	_(hipMemcpy(dst, src, sz, hipMemcpyHostToDevice));
}

void CudaContext::copyToHost(void *dst, void *src, size_t sz) {
	_(hipMemcpy(dst, src, sz, hipMemcpyDeviceToHost));
}

void CudaContext::computeRhs(REAL *f, REAL *Af, REAL *b) {
	dim3 grid(meshdata->nPlow, meshdata->nPhigh);
	dim3 block(angdata->slm); /* no need of extra threads in block */
	rightHandSide<<<grid, block>>>(*reinterpret_cast<const DeviceMeshDataRaw *>(meshdata), *reinterpret_cast<const DeviceAngularDataRaw *>(angdata), b);
	_(/*rightHandSide*/hipDeviceSynchronize());
	volumePart<<<grid, block>>>(*reinterpret_cast<const DeviceMeshDataRaw *>(meshdata), *reinterpret_cast<const DeviceAngularDataRaw *>(angdata), f, Af);
	_(/*volumePart*/hipDeviceSynchronize());
	surfacePart<<<grid, block>>>(*reinterpret_cast<const DeviceMeshDataRaw *>(meshdata), *reinterpret_cast<const DeviceAngularDataRaw *>(angdata), f, Af);
	_(/*surfacePart*/hipDeviceSynchronize());
}

__global__ void addProdKern(idx nP, idx aslm, REAL *x, const REAL *y, const REAL wy) {
	int lm = threadIdx.x;
	int vertex = blockIdx.x + blockIdx.y * gridDim.x;

	int i = vertex * aslm + lm;

	if (vertex < nP)
		x[i] += wy * y[i];
}

__global__ void mulAddKern(idx nP, idx aslm, REAL *x, const REAL wx, const REAL *y) {
	int lm = threadIdx.x;
	int vertex = blockIdx.x + blockIdx.y * gridDim.x;

	int i = vertex * aslm + lm;

	if (vertex < nP)
		x[i] = wx*x[i]+y[i];
}

__global__ void mulAddProdKern(idx nP, idx aslm, REAL *x, const REAL wx, const REAL *y, const REAL wy) {
	int lm = threadIdx.x;
	int vertex = blockIdx.x + blockIdx.y * gridDim.x;

	int i = vertex * aslm + lm;

	if (vertex < nP)
		x[i] = wx*x[i]+wy*y[i];
}

/* x += wy*y */
void CudaContext::addProd(REAL *x, const REAL *y, const REAL wy) {
	dim3 grid(meshdata->nPlow, meshdata->nPhigh);
	dim3 block(angdata->slm); /* no need of extra threads in block */
	addProdKern<<<grid, block>>>(meshdata->nP, angdata->aslm, x, y, wy);
	_(/*addProd*/hipDeviceSynchronize());
}

/* x = wx*x + y */
void CudaContext::mulAdd(REAL *x, const REAL wx, const REAL *y) {
	dim3 grid(meshdata->nPlow, meshdata->nPhigh);
	dim3 block(angdata->slm); /* no need of extra threads in block */
	mulAddKern<<<grid, block>>>(meshdata->nP, angdata->aslm, x, wx, y);
	_(/*mulAdd*/hipDeviceSynchronize());
}

void CudaContext::mulAddProd(REAL *x, const REAL wx, const REAL *y, const REAL wy) {
	dim3 grid(meshdata->nPlow, meshdata->nPhigh);
	dim3 block(angdata->slm); /* no need of extra threads in block */
	mulAddProdKern<<<grid, block>>>(meshdata->nP, angdata->aslm, x, wx, y, wy);
	_(/*mullAddProd*/hipDeviceSynchronize());
}

__global__ void normKern(idx nP, idx aslm, idx slm, REAL *x, REAL *res) {
	__shared__ REAL reduce[ASLM_MAX];
	idx lm = threadIdx.x;

	reduce[lm] = 0;
	if (lm < slm) {
		for (idx i = lm, j = nP*aslm; i < j; i += aslm) {
			REAL q = x[i];
			reduce[lm] += q*q;
		}
	} 
	__syncthreads();
#pragma unroll
	for (idx s = ASLM_MAX >> 1; s > 0; s>>=1) {
		if (lm < s)
			reduce[lm] += reduce[lm + s];
		__syncthreads();
	}
	if (lm == 0)
		res[0] = reduce[0];
}

REAL CudaContext::norm(REAL *x) {
	dim3 grid(1, 1);
	dim3 block(ASLM_MAX);
	normKern<<<grid, block>>>(meshdata->nP, angdata->slm, angdata->aslm, x, red);
	REAL hred;
	copyToHost(&hred, red, sizeof(REAL));
	hred /= meshdata->nP * angdata->slm;
	return sqrt(hred);
}	
