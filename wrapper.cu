#include "kernels.h"
#include "kernels.cu"

void *deviceAlloc(size_t size) {
	void *ret;
	hipMalloc(&ret, size);
	return ret;
}

void deviceFree(void *mem) {
	hipFree(mem);
}

void copyToDev(void *dst, void *src, size_t sz) {
	hipMemcpy(dst, src, sz, hipMemcpyHostToDevice);
}

void copyToHost(void *dst, void *src, size_t sz) {
	hipMemcpy(dst, src, sz, hipMemcpyDeviceToHost);
}

void computeRhs(const DeviceMeshData *meshdata, const DeviceAngularData *angdata, REAL *f, REAL *Af, REAL *b) {
	dim3 grid(meshdata->nPlow, meshdata->nPhigh);
	dim3 block(angdata->aslm);
	volumePart<<<grid, block>>>(*reinterpret_cast<const DeviceMeshDataRaw *>(meshdata), *reinterpret_cast<const DeviceAngularDataRaw *>(angdata), f, Af);
	surfacePart<<<grid, block>>>(*reinterpret_cast<const DeviceMeshDataRaw *>(meshdata), *reinterpret_cast<const DeviceAngularDataRaw *>(angdata), f, Af);
	rightHandSide<<<grid, block>>>(*reinterpret_cast<const DeviceMeshDataRaw *>(meshdata), *reinterpret_cast<const DeviceAngularDataRaw *>(angdata), b);
}

