#include "kernels.h"

#include "kernels.cu"

#ifdef __cplusplus
extern "C" {
#endif

void *deviceAlloc(size_t size) {
	void *ret;
	hipMalloc(&ret, size);
	return ret;
}

void deviceFree(void *mem) {
	hipFree(mem);
}

void computeRhs(MeshData *meshdata, AngularData *angdata, REAL *f, REAL *r) {
	dim3 grid(meshdata->nPlow, meshdata->nPhigh);
	dim3 block(align_power(angdata->slm, COALESCED_NUM(REAL)));
	volumePart<<<grid, block>>>(meshdata->nPlow * meshdata->nPhigh, meshdata->tetstart, meshdata->tetidx, meshdata->tetpos, meshdata->mesh, 
		angdata->slm, angdata->omega, angdata->omega_pos, f, r);
	surfacePart<<<grid, block>>>(meshdata->nPlow * meshdata->nPhigh, meshdata->facestart, meshdata->faceidx, meshdata->facepos, meshdata->bnd, 
		angdata->slm, angdata->Ox, angdata->Oy, angdata->Oz, f, r);
}

#ifdef __cplusplus
}
#endif
