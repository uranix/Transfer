#include "kernels.h"
#include "kernels.cu"
#include <stdio.h>

#ifndef _
#define _(x) do { \
	fprintf(stderr, "File %s line %d, %s is going to fail ...\n", __FILE__, __LINE__, #x); \
	fflush(stderr); \
	(x); \
	fprintf(stderr, "File %s line %d, %s failed with error `%s'\n", __FILE__, __LINE__, #x, hipGetErrorString(hipGetLastError())); \
	fflush(stderr); \
} while (0)
#endif

void *deviceAlloc(size_t size) {
	void *ret;
	_(hipMalloc(&ret, size));
	return ret;
}

void deviceFree(void *mem) {
	_(hipFree(mem));
}

void copyToDev(void *dst, void *src, size_t sz) {
	_(hipMemcpy(dst, src, sz, hipMemcpyHostToDevice));
}

void copyToHost(void *dst, void *src, size_t sz) {
	_(hipMemcpy(dst, src, sz, hipMemcpyDeviceToHost));
}

void computeRhs(const DeviceMeshData *meshdata, const DeviceAngularData *angdata, REAL *f, REAL *Af, REAL *b) {
	dim3 grid(meshdata->nPlow, meshdata->nPhigh);
	dim3 block(angdata->slm); /* no need of extra threads in block */
	volumePart<<<grid, block>>>(*reinterpret_cast<const DeviceMeshDataRaw *>(meshdata), *reinterpret_cast<const DeviceAngularDataRaw *>(angdata), f, Af);
	_(hipDeviceSynchronize());
	surfacePart<<<grid, block>>>(*reinterpret_cast<const DeviceMeshDataRaw *>(meshdata), *reinterpret_cast<const DeviceAngularDataRaw *>(angdata), f, Af);
	_(hipDeviceSynchronize());
	rightHandSide<<<grid, block>>>(*reinterpret_cast<const DeviceMeshDataRaw *>(meshdata), *reinterpret_cast<const DeviceAngularDataRaw *>(angdata), b);
	_(hipDeviceSynchronize());
}

