#include "hip/hip_runtime.h"
#include "CudaContext.h"
#include <stdio.h>
#include <math.h>

#ifndef _
#define _(x) do { \
	if ((x) != hipSuccess) { \
	fprintf(stderr, "%s:%d: %s failed with error `%s'\n", __FILE__, __LINE__, \
		#x, hipGetErrorString(hipGetLastError())); \
	fflush(stderr); } \
} while (0)
#endif

#include "kernels.cu"

void CudaContext::setDevice(int dev) {
	_(hipSetDevice(dev));
}

void *CudaContext::deviceAlloc(size_t size) {
	void *ret;
	_(hipMalloc(&ret, size));
	_(hipMemset(ret, 0, size));
	return ret;
}

void CudaContext::deviceFree(void *mem) {
	_(hipFree(mem));
}

void CudaContext::copyToDev(void *dst, void *src, size_t sz) {
	_(hipMemcpy(dst, src, sz, hipMemcpyHostToDevice));
}

void CudaContext::copyToHost(void *dst, void *src, size_t sz) {
	_(hipMemcpy(dst, src, sz, hipMemcpyDeviceToHost));
}

REAL *CudaContext::getHostSmall(void *src) {
	REAL *p = (REAL *)malloc(angdata->slm * meshdata->nP * sizeof(REAL)), 
		 *q = (REAL *)malloc(angdata->aslm * meshdata->nP * sizeof(REAL));
	_(hipMemcpy(q, src, angdata->aslm * meshdata->nP * sizeof(REAL), hipMemcpyDeviceToHost));
	for (idx i = 0; i < meshdata->nP; i++)
		for (idx j = 0; j < angdata->aslm; j++)
			if (j < angdata->slm)
				p[i * angdata->slm + j] = 
					q[i * angdata->aslm + j];
	
	free(q);
	return p;
}

void CudaContext::computeRhs(REAL *b) {
	dim3 grid(meshdata->nPlow, meshdata->nPhigh);
	dim3 block(angdata->aslm); 
	rightHandSide<<<grid, block>>>(
			*reinterpret_cast<const DeviceMeshDataRaw *>(meshdata), 
			*reinterpret_cast<const DeviceAngularDataRaw *>(angdata), b);
	_(/*rightHandSide*/hipDeviceSynchronize());
}

void CudaContext::computeLhs(REAL *f, REAL *Af) {
	dim3 grid(meshdata->nPlow, meshdata->nPhigh);
	dim3 block(angdata->aslm); 
	volumePart<<<grid, block>>>(
			*reinterpret_cast<const DeviceMeshDataRaw *>(meshdata), 
			*reinterpret_cast<const DeviceAngularDataRaw *>(angdata), f, Af);
	_(/*volumePart*/hipDeviceSynchronize());
	surfacePart<<<grid, block>>>(
			*reinterpret_cast<const DeviceMeshDataRaw *>(meshdata), 
			*reinterpret_cast<const DeviceAngularDataRaw *>(angdata), f, Af);
	_(/*surfacePart*/hipDeviceSynchronize());
}

/* x += wy*y */
void CudaContext::addProd(REAL *x, const REAL *y, const REAL wy) {
	dim3 grid(meshdata->nPlow, meshdata->nPhigh);
	dim3 block(angdata->aslm); 
	addProdKern<<<grid, block>>>(meshdata->nP, angdata->aslm, x, y, wy);
	_(/*addProd*/hipDeviceSynchronize());
}

/* x = wx*x + y */
void CudaContext::mulAdd(REAL *x, const REAL wx, const REAL *y) {
	dim3 grid(meshdata->nPlow, meshdata->nPhigh);
	dim3 block(angdata->aslm); 
	mulAddKern<<<grid, block>>>(meshdata->nP, angdata->aslm, x, wx, y);
	_(/*mulAdd*/hipDeviceSynchronize());
}

void CudaContext::mulAddProd(REAL *x, const REAL wx, const REAL *y, const REAL wy) {
	dim3 grid(meshdata->nPlow, meshdata->nPhigh);
	dim3 block(angdata->aslm); 
	mulAddProdKern<<<grid, block>>>(meshdata->nP, angdata->aslm, x, wx, y, wy);
	_(/*mullAddProd*/hipDeviceSynchronize());
}

REAL CudaContext::norm(const REAL *x) {
	dim3 grid(1, 1);
	dim3 block(ASLM_MAX);
	normKern<<<grid, block>>>(meshdata->nP, angdata->aslm, angdata->slm, x, red);
	REAL hred;
	copyToHost(&hred, red, sizeof(REAL));
	hred /= meshdata->nP * angdata->slm;
	return sqrt(hred);
}	

REAL CudaContext::dot(const REAL *x, const REAL *y) {
	dim3 grid(1, 1);
	dim3 block(ASLM_MAX);
	dotKern<<<grid, block>>>(meshdata->nP, angdata->aslm, angdata->slm, x, y, red);
	REAL hred;
	copyToHost(&hred, red, sizeof(REAL));
	return hred;
}	
