#include "hip/hip_runtime.h"
#include "util.h"

#include "MeshData.h"
#include "AngularData.h"

#include "common.cuh"

/*
Computes r = (e_i, f) + (1/kappa nabla e_i, 1/kappa nabla f) - (e_i, I_p)
	nP						: total vertices number
	start[nP+1]				: start[i+1] - start[i] = number of tetrahedrons incidental to vertex i
	idx[start[nP]]			: corresponding tetrahedron idx
	pos[start[nP]]			: local vertex idx in tetradedron
	mesh[nT]				: mesh
	slm						: anglar harmonics total number. aslm = align_power(slm, COALESCED_NUM(REAL))
	omega[3*aslm*aslm]		: values part of <Omega_i Omega_j>_lm^lms. Symmetrical to i <-> j, lm <-> lms.
							omega[i][j][lm][lms] = omega[3*aslm*aslm + aslm*lms + lm] when j = omega_pos[i][lms][lm], otherwise 0
							- coalesced when read thread <-> lm
	omega_pos[3*aslm*aslm]	: coordinate part of <Omega_i Omega_j>_lm^lms
							omega_pos[i][lms][lm] = omega_pos[3*aslm*aslm + aslm*lms + lm]
							- coalesced when read thread <-> lm
	f[aslm*nP]				: degrees of freedom 
							f[point][lm]=f[aslm*point + lm]. 
							- coalesced when reading thread <-> lm
	r[aslm*nP]				: result

Assumed:
	blockDim.x = aslm, should be less or equal ASLM_MAX.
	blockDim.y = 1
	blockDim.z = 1
	gridDim.x*gridDim.y = nP
	gridSize.z = 1

	shmem per block = 128b + 32b * blockDim.x + ? [__syncthreads()]
		~4 Kb  blockDim.x = 128
		~8 Kb blockDim.x = 256
		~32 Kb blockDim.x = 1024
   */

#define ASLM_MAX (256)

__global__ void volumePart(	DeviceMeshData md,
							DeviceAngularData ad,
							REAL *f,
							REAL *r) {
	__shared__ tetrahedron tetas;
	__shared__ REAL sums_j[4*ASLM_MAX]; /* 3 -> 4 for align*/

	idx slm = ad.slm;
	idx aslm = ad.aslm;
	REAL *omega = ad.omega;
	idx *omega_pos = ad.omega_pos;

	idx *start = md.tetstart;
	idx *tetidx = md.tetidx;
	idx *pos = md.tetpos;
	tetrahedron *mesh = md.mesh; 

	idx vertex = blockIdx.x + blockIdx.y * gridDim.x;
	idx lm = threadIdx.x;
	REAL sum_i[3];
	REAL *sum_j = &sums_j[4*lm];
	idx lo, hi;
	REAL fl[4];
	REAL fc;

	REAL sum = 0;
	lo = start[vertex];
	hi = start[vertex+1];
	for (int j = lo; j < hi; j++) {
		__syncthreads();
		tetrahedron *tet = &tetas;
		copy_unit *dst = (copy_unit *)tet;
		copy_unit *src = (copy_unit *)(mesh + tetidx[j]);
		idx local = pos[j];
		
		idx copy_incr = blockDim.x;
		for (int s=0, smax = sizeof(tetrahedron); s < smax; s += sizeof(copy_unit)*copy_incr, dst += copy_incr, src += copy_incr)
			if (s + sizeof(copy_unit) * threadIdx.x < smax)
				dst[threadIdx.x] = src[threadIdx.x];
		__syncthreads();
		fc = 0;
		#pragma unroll
		for (int s = 0; s < 4; s++) {
			REAL tmp = f[aslm*tet->p[s] + lm];
			fl[s] = tmp; 
			fc += tmp;
		}
		sum += tet->kappa_volume * (fl[local] + fc) * (1. / 20.);
		if (lm == 0)
			sum -= tet->kappa_volume * tet->I_p * (1. / 4.);
		#pragma unroll
		for (int si = 0; si < 3; si++)
			sum_i[si] = tet->s[local][si] / tet->kappa_volume * (1. / 9.); 
		#pragma unroll
		for (int sj = 0; sj < 3; sj++) {
			sum_j[sj] = 0;
			#pragma unroll
			for (int k = 0; k<4; k++)
				sum_j[sj] += fl[k] * tet->s[k][sj];
		}
		__syncthreads();
		REAL rowsum;
		idx v = lm;
		#pragma unroll
		for (int row = 0; row < 3; row++) {
			rowsum = 0;
			for (int lms = 0; lms < slm; lms++, v += aslm) {
				rowsum += omega[v] * sums_j[4*lms + omega_pos[v]];
			}
			v += (aslm-slm)*aslm;
			sum += rowsum * sum_i[row];
		}
	}
	r[aslm*vertex + lm] = sum;
}

/*
WORKS ONLY IF NORMAL IS (+/-1,0,0), (0,+/-1,0) or (0,0,+/-1).
Computes r += int_{dG x 4pi} |Omega n(x)| e_i f d Omega dS
	nP						: total vertices number
	start[nP+1]				: start[i+1] - start[i] = number of faces incidental to vertex i
	idx[start[nP]]			: corresponding face idx
	pos[start[nP]]			: local vertex idx in face
	bnd[nF]					: boundary faces
	slm						: anglar harmonics total number. aslm = align_power(slm, COALESCED_NUM(REAL))
	Ox,Oy,Oz[aslm*aslm]		: <|Omega_x|>, <|Omega_y|>, <|Omega_z|>.
	f[aslm*nP]				: degrees of freedom 
							f[point][lm]=f[aslm*point + lm]. 
							- coalesced when reading thread <-> lm
	r[aslm*nP]				: result

Assumed:
	blockDim.x = aslm, should be less or equal ASLM_MAX.
	blockDim.y = 1
	blockDim.z = 1
	gridDim.x*gridDim.y = nP
	gridSize.z = 1

	shmem per block = 32*ASLM_MAX * blockDim.x + ? [__syncthreads()]
   */
__global__ void surfacePart( DeviceMeshData md,
							 DeviceAngularData ad,
							 REAL *f, 
							 REAL *r) 
{
	__shared__ REAL fv[4*ASLM_MAX]; /* f1,f2,f3,fc */
	REAL *On;
	face triangle;

	idx *start = md.facestart;
	idx *faceidx = md.faceidx;
	idx *pos = md.facepos;
	face *bnd = md.bnd;

	idx slm = ad.slm; 
	idx aslm = ad.aslm;
	REAL *Ox = ad.Ox;
	REAL *Oy = ad.Oy;
	REAL *Oz = ad.Oz;

	idx vertex = blockIdx.x + blockIdx.y * gridDim.x;
	idx lm = threadIdx.x;
	idx lo, hi;

	REAL sum = 0;
	lo = start[vertex];
	hi = start[vertex+1];
	for (int j = lo; j < hi; j++) {
		__syncthreads();
		face *tr = &triangle;
		copy_unit *dst = (copy_unit *)tr;
		copy_unit *src = (copy_unit *)(bnd + faceidx[j]);
		idx local = pos[j];
		
		idx copy_incr = blockDim.x;
		for (int s=0, smax = sizeof(face); s < smax; s += sizeof(copy_unit)*copy_incr, dst += copy_incr, src += copy_incr)
			if (s + sizeof(copy_unit) * threadIdx.x < smax)
				dst[threadIdx.x] = src[threadIdx.x];
		__syncthreads();
		REAL surf = abs(tr->s[0] + tr->s[1] + tr->s[2]);
		if (2*abs(tr->s[0]) > surf)
			On = Ox;
		else if (2*abs(tr->s[1]) > surf)
			On = Oy;
		else
			On = Oz;
		fv[3*aslm+lm] = 0;
		#pragma unroll
		for (int s = 0; s<3; s++) {
			REAL tmp = f[aslm*tr->p[s]+lm];
			fv[s*aslm+lm] = tmp;
			fv[3*aslm+lm] += tmp;
		}
		__syncthreads();
		for (int lms = 0; lms < slm; lms ++)
			sum += surf * On[aslm * lms + lm] * (fv[local*aslm + slm] + fv[3*aslm + slm]) * (1. / 12.);
	}
	r[aslm*vertex + lm] += sum;
}
