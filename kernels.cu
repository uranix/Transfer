#include "hip/hip_runtime.h"
#include "util.h"

#include "MeshData.h"
#include "AngularData.h"

#include "common.cuh"

/*
   sz must be multiple of sizeof(copy_unit);
   dst and src should be aligned of sizeof(copy_unit) boundary
 */
__device__ void dmemcpy(void *_dst, const void *_src, idx sz) {
	copy_unit *dst = (copy_unit *)_dst;
	copy_unit *src = (copy_unit *)_src;
	for (idx s = threadIdx.x, smax = sz / sizeof(copy_unit); s < smax; s++)
		dst[s] = src[s];
}

/*
Computes right-hand size of SLAE r = (e_i, I_p)
 */

__global__ void rightHandSide(	DeviceMeshDataRaw md,
								REAL *r)
{
	__shared__ tetrahedron stet;
	tetrahedron *tet = &stet;

	idx *start = md.tetstart;
	idx *tetidx = md.tetidx;
	idx *pos = md.tetpos;
	tetrahedron *mesh = md.mesh; 

	idx vertex = blockIdx.x + blockIdx.y * gridDim.x;
	idx lm = threadIdx.x;
	idx lo, hi;

	REAL sum = 0;
	lo = start[vertex];
	hi = start[vertex+1];
	for (int j = lo; j < hi; j++) {
		__syncthreads();
		idx local = pos[j];
		
		dmemcpy(tet, mesh+tetidx[j], sizeof(tetrahedron));
		__syncthreads();
		if (lm == 0)
			sum += tet->kappa_volume * tet->I_p * (1. / 4.);
	}
	if (vertex < md.nP)
		r[aslm*vertex + lm] = sum;
}

/*
Computes r = (e_i, f) + (1/kappa nabla e_i, 1/kappa nabla f)

	nP						: total vertices number
	start[nP+1]				: start[i+1] - start[i] = number of tetrahedrons incidental to vertex i
	idx[start[nP]]			: corresponding tetrahedron idx
	pos[start[nP]]			: local vertex idx in tetradedron
	mesh[nT]				: mesh
	slm						: angular harmonics total number. aslm = align_power(slm, COALESCED_NUM(REAL))
	omega[3*aslm*aslm]		: values part of <Omega_i Omega_j>_lm^lms. Symmetrical to i <-> j, lm <-> lms.
							omega[i][j][lm][lms] = omega[i*aslm*aslm + aslm*lms + lm] when j = omega_pos[i][lms][lm], otherwise 0
							- coalesced when read thread <-> lm
	omega_pos[3*aslm*aslm]	: coordinate part of <Omega_i Omega_j>_lm^lms
							omega_pos[i][lms][lm] = omega_pos[3*aslm*aslm + aslm*lms + lm]
							- coalesced when read thread <-> lm
	f[aslm*nP]				: degrees of freedom 
							f[point][lm]=f[aslm*point + lm]. 
							- coalesced when reading thread <-> lm
	r[aslm*nP]				: result

Assumed:
	blockDim.x = aslm, should be less or equal ASLM_MAX.
	blockDim.y = 1
	blockDim.z = 1
	gridDim.x*gridDim.y = nP
	gridSize.z = 1

	shmem per block = 128b + 24b * blockDim.x + ? [__syncthreads()]
		~4 Kb  blockDim.x = 128
		~8 Kb blockDim.x = 256
		~32 Kb blockDim.x = 1024
   */

#define ASLM_MAX (256)

__global__ void volumePart(	DeviceMeshDataRaw md,
							DeviceAngularDataRaw ad,
							REAL *f,
							REAL *r) 
{
	__shared__ tetrahedron stet;
	__shared__ REAL sums_j[3*ASLM_MAX]; 
	tetrahedron *tet = &stet;

	idx slm = ad.slm;
	idx aslm = ad.aslm;
	REAL *omega = ad.omega;
	idx *omega_pos = ad.omega_pos;

	idx *start = md.tetstart;
	idx *tetidx = md.tetidx;
	idx *pos = md.tetpos;
	tetrahedron *mesh = md.mesh; 

	idx vertex = blockIdx.x + blockIdx.y * gridDim.x;
	idx lm = threadIdx.x;
	REAL sum_i[3];
	idx lo, hi;
	REAL fl[4];
	REAL fsum;

	REAL sum = 0;
	lo = start[vertex];
	hi = start[vertex+1];
	for (int j = lo; j < hi; j++) {
		__syncthreads();
		idx local = pos[j];
		
		dmemcpy(tet, mesh+tetidx[j], sizeof(tetrahedron));
		__syncthreads();
		fc = 0;
		#pragma unroll
		for (int s = 0; s < 4; s++) {
			REAL tmp = f[aslm*tet->p[s] + lm];
			fl[s] = tmp; 
			fsum += tmp;
		}
		sum += tet->kappa_volume * (fl[local] + fsum) * (1. / 20.);
		#pragma unroll
		for (int si = 0; si < 3; si++)
			sum_i[si] = tet->s[local][si] / tet->kappa_volume * (1. / 9.); 
		#pragma unroll
		for (idx sj = 0, v = lm; sj < 3; sj++, v += aslm) {
			sums_j[v] = 0;
			#pragma unroll
			for (int k = 0; k<4; k++)
				sums_j[v] += fl[k] * tet->s[k][sj];
		}
		__syncthreads();
		REAL rowsum;
		/* nvcc can't unroll this.
		idx v = lm;
		#pragma unroll
		for (int row = 0; row < 3; row++) {
			rowsum = 0;
			for (int lms = 0; lms < slm; lms++, v += aslm) {
				rowsum += omega[v] * sums_j[omega_pos[v]*aslm + lms];
			}
			v += (aslm-slm)*aslm;
			sum += rowsum * sum_i[row];
		}
		*/
		rowsum = 0;
		for (idx lms = 0, v = lm; lms < slm; lms++, v += aslm)
			rowsum += omega[v] * sums_j[omega_pos[v]*aslm + lms]; /* TODO: resolve bank conflict (x16) */
		sum += rowsum * sum_i[0];

		rowsum = 0;
		for (idx lms = 0, v = lm + aslm*aslm; lms < slm; lms++, v += aslm)
			rowsum += omega[v] * sums_j[omega_pos[v]*aslm + lms]; /* TODO: resolve bank conflict (x16) */
		sum += rowsum * sum_i[1];

		rowsum = 0;
		for (idx lms = 0, v = lm + aslm*aslm*2; lms < slm; lms++, v += aslm)
			rowsum += omega[v] * sums_j[omega_pos[v]*aslm + lms]; /* TODO: resolve bank conflict (x16) */
		sum += rowsum * sum_i[2];
	}
	if (vertex < md.nP)
		r[aslm*vertex + lm] = sum;
}

/*
WORKS ONLY IF NORMAL IS (+/-1,0,0), (0,+/-1,0) or (0,0,+/-1). Issue #15
Computes r += int_{dG x 4pi} |Omega n(x)| e_i f d Omega dS
	nP						: total vertices number
	start[nP+1]				: start[i+1] - start[i] = number of faces incidental to vertex i
	idx[start[nP]]			: corresponding face idx
	pos[start[nP]]			: local vertex idx in face
	bnd[nF]					: boundary faces
	slm						: angular harmonics total number. aslm = align_power(slm, COALESCED_NUM(REAL))
	Ox,Oy,Oz[aslm*aslm]		: <|Omega_x|>, <|Omega_y|>, <|Omega_z|>.
	f[aslm*nP]				: degrees of freedom 
							f[point][lm]=f[aslm*point + lm]. 
							- coalesced when reading thread <-> lm
	r[aslm*nP]				: result

Assumed:
	blockDim.x = aslm, should be less or equal ASLM_MAX.
	blockDim.y = 1
	blockDim.z = 1
	gridDim.x*gridDim.y = nP
	gridSize.z = 1

	shmem per block = 32*ASLM_MAX * blockDim.x + ? [__syncthreads()]
   */
__global__ void surfacePart( DeviceMeshDataRaw md,
							 DeviceAngularDataRaw ad,
							 REAL *f, 
							 REAL *r) 
{
	__shared__ REAL fv[4*ASLM_MAX]; /* f1,f2,f3,fsum */
	__shared__ face stri;
	face *tr = &stri;
	REAL *On;

	idx *start = md.facestart;
	idx *faceidx = md.faceidx;
	idx *pos = md.facepos;
	face *bnd = md.bnd;

	idx slm = ad.slm; 
	idx aslm = ad.aslm;
	REAL *Ox = ad.Ox;
	REAL *Oy = ad.Oy;
	REAL *Oz = ad.Oz;

	idx vertex = blockIdx.x + blockIdx.y * gridDim.x;
	idx lm = threadIdx.x;
	idx lo, hi;

	REAL sum = 0;
	lo = start[vertex];
	hi = start[vertex+1];
	for (int j = lo; j < hi; j++) {
		__syncthreads();
		idx local = pos[j];	
		dmemcpy(tr, bnd + faceidx[j], sizeof(tr));
		__syncthreads();
		REAL surf = abs(tr->s[0] + tr->s[1] + tr->s[2]);
		if (2*abs(tr->s[0]) > surf)
			On = Ox;
		else if (2*abs(tr->s[1]) > surf)
			On = Oy;
		else
			On = Oz;
		fv[3*aslm+lm] = 0;
		#pragma unroll
		for (int s = 0; s<3; s++) {
			REAL tmp = f[aslm*tr->p[s]+lm];
			fv[s*aslm+lm] = tmp;
			fv[3*aslm+lm] += tmp;
		}
		__syncthreads();
		for (int lms = 0; lms < slm; lms ++)
			sum += surf * On[aslm * lms + lm] * (fv[local*aslm + slm] + fv[3*aslm + slm]) * (1. / 12.);
	}
	if (vertex < md.nP)
		r[aslm*vertex + lm] += sum;
}
